#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#define _SCL_SECURE_NO_WARNINGS

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <sstream>
#include <string>
#include <time.h>
#include <fstream>

using namespace std;
using namespace thrust;

// Thrust is a C++ template library for CUDA based on the Standard Template Library (STL).
// Thrust allows you to implement high performance parallel applications with minimal programming effort through a high-level interface that is fully interoperable with CUDA C.
// Thrust provides a rich collection of data parallel primitives such as scan, sort, and reduce, which can be composed together to implement complex algorithms with concise, readable source code.
// By describing your computation in terms of these high-level abstractions you provide Thrust with the freedom to select the most efficient implementation automatically.
// As a result, Thrust can be utilized in rapid prototyping of CUDA applications, where programmer productivity matters most, as well as in production, where robustness and absolute performance are crucial.
// Read more at: http://docs.nvidia.com/cuda/thrust/index.html#ixzz3hymTnQwX 

template <typename T>
T mul_functor(T value1, T value2)
{
	return value1 * value2;
}

template <typename T>
T add_functor(T value1, T value2)
{
	return value1 + value2;
}

struct t_previous_result
{
	int m;
	int n;
	int k;
	int grid_size;
	int block_size;
	double price;
};

/////////////////////////////////////////////////////////
// ���������� ��������� ����� ����� ��������� ���������
double delta(std::vector<double> x, std::vector<double> y)
{
	auto s = 0.0;
	auto i = 0;
	for (; i < x.size() && i < y.size(); i++) s += (x[i] - y[i])*(x[i] - y[i]);
	for (; i < x.size(); i++) s += x[i] * x[i];
	for (; i < y.size(); i++) s += y[i] * y[i];
	return s;
}

/////////////////////////////////////////////////////////
// ���������� ������������ ���� ��� ��������� ���������� ������ �� ������������ ������
double predict(int grid_size, int block_size, size_t m, size_t n, size_t k, std::vector<t_previous_result> & previous_results)
{
		std::vector<double> x1;
		x1.push_back(m);
		x1.push_back(n);
		x1.push_back(k);
		x1.push_back(grid_size);
		x1.push_back(block_size);
		auto sy = 0.0;
		auto sw = 0.0;
		for (auto it = previous_results.begin(); it != previous_results.end(); ++it)
		{
			std::vector<double> x2;
			x2.push_back(it->m);
			x2.push_back(it->n);
			x2.push_back(it->k);
			x2.push_back(it->grid_size);
			x2.push_back(it->block_size);

			auto w = 1.0/(1.0+delta(x1, x2));
			sy += it->price*w;
			sw += w;
		}
		return sy / sw;
}

__global__ void matrix_mul_kernel(double *a, double *b, double *c, size_t m, size_t n, size_t k)
{
	int total = m*k;
	for (int id = threadIdx.x; id < total; id += blockDim.x)
	{
		int x = id / k;
		int y = id % k;
		double s = 0;
		for (int i = 0; i < n; i++)
			s += a[x*n + i] * b[i*k + y];
		c[id] = s;

	}
}

__global__ void matrix_fill_kernel(double *a, size_t m, size_t n)
{
	int total = m*n;
	for (int id = threadIdx.x; id < total; id += blockDim.x)
		a[id] = id;
}

/////////////////////////////////////////////////////////
// �������� ���������� � ���������� ���� ��� ��������� ����������
__host__ double body(int grid_size, int block_size, size_t m, size_t n, size_t k, int count)
{
	auto time = clock();
	for (auto s = 0; s < count; s++)
	{
		thrust::device_vector<double> a(m*n);
		thrust::device_vector<double> b(n*k);
		thrust::device_vector<double> c(m*k);
		auto aPtr = thrust::raw_pointer_cast(&a[0]);
		auto bPtr = thrust::raw_pointer_cast(&b[0]);
		auto cPtr = thrust::raw_pointer_cast(&c[0]);
		matrix_fill_kernel <<< grid_size, block_size >>> (aPtr, m, n);
		matrix_fill_kernel <<< grid_size, block_size >>> (bPtr, n, k);
		matrix_mul_kernel <<< grid_size, block_size >>> (aPtr, bPtr, cPtr, m, n, k);
	}
	time = clock() - time;
	auto price = static_cast<double>(time) / CLOCKS_PER_SEC / count / m/n/k;
	return price;
}


enum t_mode
{
	DEFAULT = 0,
	GENERATE = 1,
	PREDICT = 2,
	COMPARE = 3
};

t_mode mode = DEFAULT;

/////////////////////////////////////////////////////////
// ��������� ��������
static const unsigned _count = 1;
static const int _m = 2;
static const int _n = 2;
static const int _k = 2;
static const int _grid_size = 1;
static const int _block_size = 255;

int main(int argc, char* argv[])
{
	std::vector<t_previous_result> previous_results;

	auto count = _count;

	char* input_file_name = NULL;
	char* output_file_name = NULL;
	char* previous_results_file_name = NULL;

	auto m = _m;
	auto n = _n;
	auto k = _k;
	auto grid_size = _grid_size;
	auto block_size = _block_size;

	// ��������� ��������� � ������� Windows
	// ������� setlocale() ����� ��� ���������, ������ �������� - ��� ��������� ������, � ����� ������ LC_TYPE - ����� ��������, ������ �������� � �������� ������. 
	// ������ ������� ��������� ����� ������ "Russian", ��� ��������� ������ ������� �������, ����� ����� �������� ����� ����� �� ��� � � ��.
	setlocale(LC_ALL, "");

	for (int i = 1; i < argc; i++)
	{
		if (strcmp(argv[i], "-help") == 0)
		{
			std::cout << "Usage :\t" << argv[0] << " [...] [g <grid_size>] [b <block_size>] [-input <inputfile>] [-output <outputfile>]" << std::endl;
			std::cout << "\t-c <���������� ���������� ��������� ��� ������ �������>" << std::endl;
		}
		else if (strcmp(argv[i], "-generate") == 0) mode = GENERATE;
		else if (strcmp(argv[i], "-predict") == 0) mode = PREDICT;
		else if (strcmp(argv[i], "-compare") == 0) mode = COMPARE;
		else if (strcmp(argv[i], "-c") == 0) count = atoi(argv[++i]);
		else if (strcmp(argv[i], "-input") == 0) input_file_name = argv[++i];
		else if (strcmp(argv[i], "-output") == 0) output_file_name = argv[++i];
		else if (strcmp(argv[i], "-history") == 0) previous_results_file_name = argv[++i];
		else if (strcmp(argv[i], "m") == 0) m = atoi(argv[++i]);
		else if (strcmp(argv[i], "n") == 0) n = atoi(argv[++i]);
		else if (strcmp(argv[i], "k") == 0) k = atoi(argv[++i]);
		else if (strcmp(argv[i], "g") == 0) grid_size = atoi(argv[++i]);
		else if (strcmp(argv[i], "b") == 0) block_size = atoi(argv[++i]);
	}

	if (input_file_name != NULL) freopen(input_file_name, "r", stdin);
	if (output_file_name != NULL) freopen(output_file_name, "w", stdout);

	if (previous_results_file_name != NULL)
	{
		std::ifstream history(previous_results_file_name);
		if (!history.is_open()) throw "Error opening file";
		std::string line;
		while (std::getline(history, line))
		{
			std::stringstream lineStream(line);
			int m;
			int n;
			int k;
			int grid_size;
			int block_size;
			double price;
			lineStream >> m >> n >> k >> grid_size >> block_size >> price;
			t_previous_result previous_result;
			previous_result.m = m;
			previous_result.n = n;
			previous_result.k = k;
			previous_result.grid_size = grid_size;
			previous_result.block_size = block_size;
			previous_result.price = price;
			previous_results.push_back(previous_result);
		}
	}

		std::string line;
		switch (mode)
		{
		case GENERATE:
			for (std::getline(std::cin, line); !line.empty(); std::getline(std::cin, line))
			{
				std::stringstream lineStream(line);
				lineStream >> m >> n >> k >> grid_size >> block_size;
				auto price = body(grid_size, block_size, m, n, k, count);
				std::cout << m << ' ' << n << ' ' << k << ' ' << grid_size << ' ' << block_size << ' ' << price << std::endl;
			}
			break;
		case PREDICT:
			for (std::getline(std::cin, line); !line.empty(); std::getline(std::cin, line))
			{
				std::stringstream lineStream(line);
				lineStream >> m >> n >> k >> grid_size >> block_size;
				auto price = predict(grid_size, block_size, m, n, k, previous_results);
				std::cout << m << ' ' << n << ' ' << k << ' ' << grid_size << ' ' << block_size << ' ' << price << std::endl;
			}
			break;
		case COMPARE:
			for (std::getline(std::cin, line); !line.empty(); std::getline(std::cin, line))
			{
				std::stringstream lineStream(line);
				lineStream >> m >> n >> k >> grid_size >> block_size;
				int grid_size1;
				int block_size1;
				// ��������� ���������� � ����������� �����������
				auto price = body(grid_size, block_size, m, n, k, count);
				// ������� ��������� ��� ������� ������������� ���� ����� ����������
				double price1 = DBL_MAX;
				for (auto i = 1; i < 32; i++)
					for (auto j = 1; j < 256; j++)
					{
						auto price = predict(grid_size, block_size, m, n, k, previous_results);
						if (price1 < price) continue;
						grid_size1 = i;
						block_size1 = j;
						price1 = price;
					}
				// ��������� ���������� � ����������� ��� ������� ������������� ���� ����������
				price1 = body(grid_size1, block_size1, m, n, k, count);
				std::cout << m << ' ' << n << ' ' << k << ' ' 
					<< grid_size << '/' << grid_size1 << ' ' 
					<< block_size << '/' << block_size1 << ' ' 
					<< price << '/' << price1 << std::endl;
			}
			break;
		default:
			{
				auto price = body(grid_size, block_size, m, n, k, count);
				std::cout << m << ' ' << n << ' ' << k << ' ' << grid_size << ' ' << block_size << ' ' << price << std::endl;
			}
			break;
		}

	return 0;
}

